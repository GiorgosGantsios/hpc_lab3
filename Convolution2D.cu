/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	5
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            goto cleanup; \
        } \
    } while (0)
 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }    
}

__global__ void convolutionRowGPU(double *d_Dst, double *d_Src, double *d_Filter, 
  int imageW, int imageH, int filterR) {
  
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int y = index / imageW; // row
  int x = index % imageW; // col
  double sum = 0;
  // printf("Blockidx.x: %d, BlockDim.x: %d, threadIdx.x: %d\n", blockIdx.x, blockDim.x, threadIdx.x);
  if(x < imageW && y < imageH){
    for(int k = -filterR; k <= filterR; k++){
      int d = x + k;
      if(d >= 0 && d < imageW)
        sum += d_Src[y * imageW + d] * d_Filter[filterR - k];
    }
    d_Dst[index] = sum;
  }
}

__global__ void convolutionColGPU(double *d_Dst, double *d_Src, double *d_Filter, 
  int imageW, int imageH, int filterR) {
  
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int y = index / imageW; // row
  int x = index % imageW; // col
  double sum = 0;
  if(x < imageW && y < imageH){
    for(int k = -filterR; k <= filterR; k++){
      int d = y + k;
      if(d >= 0 && d < imageH)
        sum += d_Src[d * imageW + x] * d_Filter[filterR - k];
    }
    d_Dst[index] = sum;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}




////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    double
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;
    double *d_Filter = NULL;
    double*d_Input = NULL;
    double *d_Buffer = NULL;
    double *d_Output = NULL;

    int imageW;
    int imageH;
    int extra_block;  // used to indicate if an extra block is needed (if image size is not divisible by 1024)
    unsigned int i;

    // CUDA Error Checking
    hipError_t e;

    // CPU time measuring variables
    clock_t start, end;

    // CUDA measuring events
    hipEvent_t startCuda, stopCuda;
    float millisecondsTransfers = 0, millisecondsKernelsandTransferBack = 0;

	  printf("Enter filter radius : ");
	  // scanf("%d", &filter_radius);
    if(!scanf("%d", &filter_radius)){
      printf("ERROR: scanf: FILE: %s, LINE: %d\n", __FILE__, __LINE__);
    }

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    // scanf("%d", &imageW);
    if(!scanf("%d", &imageW)){
      printf("ERROR: scanf: FILE: %s, LINE: %d\n", __FILE__, __LINE__);
    }
    imageH = imageW;
    extra_block = ((imageH*imageW)%1024 != 0);

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
    h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputGPU = (double *)malloc(imageW * imageH * sizeof(double));

    if(!(h_Filter && h_Input && h_Buffer && h_OutputCPU && h_OutputGPU)){
      printf("ERROR on Host Allocation!\n");
      goto cleanup;
    }
    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
    }

    printf("Allocating GPU memory...\n");

    CHECK_CUDA_ERROR(hipMalloc((void **)&d_Filter, FILTER_LENGTH * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_Input, imageW * imageH * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_Buffer, imageW * imageH * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_Output, imageW * imageH * sizeof(double)));


    // Start Measuring memory transfer times
    hipEventCreate(&startCuda);
    hipEventCreate(&stopCuda);

    hipEventRecord(startCuda, 0);

    CHECK_CUDA_ERROR(hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(double), hipMemcpyHostToDevice));

    hipEventRecord(stopCuda, 0);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    start = clock();
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    end = clock();
    printf("CPU time: %f msec\n", (double)((end - start)/(CLOCKS_PER_SEC/1000000))/1000);
    printf("GPU computation...\n");

    // 1st Kernel launch
    // No need for sync barrier because cudaMemCpy(..., hostToDevice) works as barrier
    //printf("1st Kernel Launch: Row Conolution: ");
    hipEventRecord(startCuda, 0 );
    convolutionRowGPU<<<(imageH*imageW)/1024+extra_block, 1024>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    e = hipGetLastError();
    if(e!=hipSuccess){
      printf("ERROR: %s, FILE: %s, LINE: %d\n", hipGetErrorString(e), __FILE__, __LINE__);
      goto cleanup;
    }
    //else{
    //  printf("cudaGetLastError() == cudaSuccess!\n");
    //}

    // 2nd Kernel launch
    // Synchronize between 2 kernels launch because Column kernel needs the d_Buffer as input.
    // d_Buffer works as intermediate result so we ensure it is completely written.

    //printf("2nd Kernel Launch: Col Conolution: ");
    convolutionColGPU<<<(imageH*imageW)/1024+extra_block, 1024>>>(d_Output, d_Buffer, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    e = hipGetLastError();
    if(e!=hipSuccess){
      printf("ERROR: %s, FILE: %s, LINE: %d\n", hipGetErrorString(e), __FILE__, __LINE__);
      goto cleanup;
    }
    //else{
    //  printf("cudaGetLastError() == cudaSuccess!\n");
    //}

    CHECK_CUDA_ERROR(hipMemcpy(h_OutputGPU, d_Output, imageW * imageH * sizeof(double), hipMemcpyDeviceToHost));
    hipEventRecord(stopCuda, 0);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsKernelsandTransferBack, startCuda, stopCuda);
    printf("GPU Time: %f msec\n", millisecondsTransfers+millisecondsKernelsandTransferBack);
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    for (i = 0; i < imageW * imageH; i++) {
        if(ABS(h_OutputGPU[i] - h_OutputCPU[i]) > accuracy){
          printf("Difference bigger than accuray in index i = %d, h_OutputGPU[%d] = %lf, h_OutputCPU[%d] = %lf\n", i, i, h_OutputGPU[i], i, h_OutputCPU[i]);
          goto cleanup;
        }
    }

cleanup:
    // free all the allocated memory
    printf("Deallocating host memory\n");
    if (h_OutputCPU) free(h_OutputCPU);
    if (h_Buffer) free(h_Buffer);
    if (h_Input) free(h_Input);
    if (h_Filter) free(h_Filter);
    if (h_OutputGPU) free(h_OutputGPU);

    printf("Deallocating device memory\n");
    if (d_Output) hipFree(d_Output);
    if (d_Buffer) hipFree(d_Buffer);
    if (d_Input) hipFree(d_Input);
    if (d_Filter) hipFree(d_Filter);

    printf("Destroying CUDA events\n");
    CHECK_CUDA_ERROR(hipEventDestroy(startCuda));
    CHECK_CUDA_ERROR(hipEventDestroy(stopCuda));
    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    printf("Reset Device: ");

    hipDeviceSynchronize();
    e = hipGetLastError();
    if(e!=hipSuccess){
      printf("ERROR: %s, FILE: %s, LINE: %d\n", hipGetErrorString(e), __FILE__, __LINE__);
    }
    else{
      printf("cudaGetLastError() == cudaSuccess!\n");
    }

    hipDeviceReset();


    return 0;
}
