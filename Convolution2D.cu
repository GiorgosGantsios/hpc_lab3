/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.1
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            goto cleanup; \
        } \
    } while (0)
 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = filterR; y < imageH+filterR; y++) {
    for (x = filterR; x < imageW+filterR; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        //if (d >= 0 && d < imageW) {
        sum += h_Src[y * (imageW+2*filterR) + d] * h_Filter[filterR - k];
        

        h_Dst[y * (imageW+2*filterR) + x] = sum;
        printf("index: %d\n", (y * (imageW+2*filterR) + x));
      }
    }
  }    
}

__global__ void convolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, 
  int imageW, int imageH, int filterR) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int y = index / imageW; // row
  int x = index % imageW; // col
  y+= filterR;
  x+= filterR;
  float sum = 0;
  // printf("Blockidx.x: %d, BlockDim.x: %d, threadIdx.x: %d\n", blockIdx.x, blockDim.x, threadIdx.x);
  //if(x < imageW && y < imageH){
    for(int k = -filterR; k <= filterR; k++){
      int d = x + k;
      //if(d >= 0 && d < imageW)
        sum += d_Src[y * (imageW+2*filterR) + d] * d_Filter[filterR - k];
    }
    //if (x >= filterR && x < (imageW + filterR) && y >= filterR && y < (imageH + filterR)) {
    // Apply filter here
    d_Dst[y * (imageW + 2 * filterR) + x] = sum;
    //printf("K1 d_Dst: %d, sum: %f, x: %d\n", (y * (imageW + 2 * filterR) + x), sum, x);
    
    //d_Dst[y * (imageW+2*filterR) + x] = sum;
  
}

__global__ void convolutionColGPU(float *d_Dst, float *d_Src, float *d_Filter, 
  int imageW, int imageH, int filterR) {
  
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int y = index / imageW; // row
  int x = index % imageW; // col
  y += filterR;
  x += filterR;
  float sum = 0;
  //if(x < imageW && y < imageH){
    for(int k = -filterR; k <= filterR; k++){
      int d = y + k;
      //if(d >= 0 && d < imageH)
        sum += d_Src[d * (imageW+2*filterR) + x] * d_Filter[filterR - k];
    }
    //if (x >= filterR && x < (imageW + filterR) && y >= filterR && y < (imageH + filterR)) {
    // Apply filter here
      d_Dst[(y-filterR) * (imageW) + (x-filterR)] = sum;
      //printf("K2 d_Dst: %d, sum: %f, x: %d\n", (y * (imageW + 2 * filterR) + x), sum, x);
    
    //d_Dst[y * (imageW+2*filterR) + x] = sum;
  
}

////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = filterR; y < imageH+filterR; y++) {
    for (x = filterR; x < imageW+filterR; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        //if (d >= 0 && d < imageH) {
          sum += h_Src[d * (imageW+2*filterR) + x] * h_Filter[filterR - k];
          
 
        h_Dst[(y-filterR) * imageW + (x-filterR)] = sum;
      }
    }
  }
    
}




////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;
    float *d_Filter = NULL;
    float*d_Input = NULL;
    float *d_Buffer = NULL;
    float *d_Output = NULL;

    int imageW;
    int imageH;
    int extra_block;  // used to indicate if an extra block is needed (if image size is not divisible by 1024)
    unsigned int i;

    hipError_t e;
    clock_t start, end;


	  printf("Enter filter radius : ");
	  // scanf("%d", &filter_radius);
    if(!scanf("%d", &filter_radius)){
      printf("ERROR: scanf: FILE: %s, LINE: %d\n", __FILE__, __LINE__);
    }

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    // scanf("%d", &imageW);
    if(!scanf("%d", &imageW)){
      printf("ERROR: scanf: FILE: %s, LINE: %d\n", __FILE__, __LINE__);
    }
    imageH = imageW;
    extra_block = ((imageH*imageW)%1024 != 0);

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)calloc((imageW+2*filter_radius) * (imageH+2*filter_radius), sizeof(float)); // use calloc to initialize padded elements to 0
    h_Buffer    = (float *)calloc((imageW+2*filter_radius) * (imageH+2*filter_radius), sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));

    if(!(h_Filter && h_Input && h_Buffer && h_OutputCPU && h_OutputGPU)){
      printf("ERROR on Host Allocation!\n");
      goto cleanup;
    }
    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

 for (i = filter_radius; i < filter_radius + imageH; i++) {
        int row = i*(2*filter_radius + imageW);
        for(int j = filter_radius; j < imageW + filter_radius; j++){
          h_Input[row+j] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
        }
        // h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }


    printf("Padded 12x12 CPU Array:\n");
    for (int i = 0; i < (imageH+2*filter_radius); i++) {
        for (int j = 0; j < (imageW+2*filter_radius); j++) {
            printf("%8.2f ", h_Input[i * (imageW+2*filter_radius) + j]);
        }
        printf("\n");
    }


    printf("Allocating GPU memory...\n");

    CHECK_CUDA_ERROR(hipMalloc((void **)&d_Filter, FILTER_LENGTH * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_Input, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_Buffer, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_Output, imageW * imageH * sizeof(float)));

    hipMemset(d_Input, 0, (imageW+2*filter_radius) * (imageH+2*filter_radius));
    hipMemset(d_Buffer, 0, (imageW+2*filter_radius) * (imageH+2*filter_radius));
    hipMemset(d_Output, 0, imageW * imageH);

    CHECK_CUDA_ERROR(hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_Input, h_Input, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(float), hipMemcpyHostToDevice));



    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    start = clock();
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    end = clock();
    printf("CPU time: %ld\n", end - start);
    printf("GPU computation...\n");

    printf("Padded 12x12 CPU Array AFTER:\n");
    for (int i = 0; i < imageH; i++) {
        for (int j = 0; j < imageW; j++) {
            printf("%8.2f ", h_OutputCPU[i * imageW + j]);
        }
        printf("\n");
    }

    // 1st Kernel launch
    // No need for sync barrier because cudaMemCpy(..., hostToDevice) works as barrier
    start = clock();
    printf("1st Kernel Launch: Row Conolution: ");
    convolutionRowGPU<<<(imageH*imageW)/1024+extra_block, 1024>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    e = hipGetLastError();
    if(e!=hipSuccess){
      printf("ERROR: %s, FILE: %s, LINE: %d\n", hipGetErrorString(e), __FILE__, __LINE__);
      goto cleanup;
    }
    else{
      printf("cudaGetLastError() == cudaSuccess!\n");
    }

    // 2nd Kernel launch
    // Synchronize between 2 kernels launch because Column kernel needs the d_Buffer as input.
    // d_Buffer works as intermediate result so we ensure it is completely written.

    printf("2nd Kernel Launch: Col Conolution: ");
    convolutionColGPU<<<(imageH*imageW)/1024+extra_block, 1024>>>(d_Output, d_Buffer, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    e = hipGetLastError();
    if(e!=hipSuccess){
      printf("ERROR: %s, FILE: %s, LINE: %d\n", hipGetErrorString(e), __FILE__, __LINE__);
      goto cleanup;
    }
    else{
      printf("cudaGetLastError() == cudaSuccess!\n");
    }
    end = clock();
    printf("GPU time: %ld\n", end - start);

    CHECK_CUDA_ERROR(hipMemcpy(h_OutputGPU, d_Output, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost));

    printf("Padded 12x12 GPU Array:\n");
    for (int i = 0; i < imageH; i++) {
        for (int j = 0; j < imageW; j++) {
            printf("%8.2f ", h_OutputGPU[i * imageW + j]);
        }
        printf("\n");
    }
    
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    for (i = 0; i < imageW * imageH; i++) {
        if(ABS(h_OutputGPU[i] - h_OutputCPU[i]) > accuracy){
          printf("Difference bigger than accuray in index i = %d, h_OutputGPU[%d] = %f, h_OutputCPU[%d] = %f\n", i, i, h_OutputGPU[i], i, h_OutputCPU[i]);
          goto cleanup;
        }
    }

cleanup:
    // free all the allocated memory
    printf("Deallocating host memory\n");
    if (h_OutputCPU) free(h_OutputCPU);
    if (h_Buffer) free(h_Buffer);
    if (h_Input) free(h_Input);
    if (h_Filter) free(h_Filter);
    if (h_OutputGPU) free(h_OutputGPU);

    printf("Deallocating device memory\n");
    if (d_Output) hipFree(d_Output);
    if (d_Buffer) hipFree(d_Buffer);
    if (d_Input) hipFree(d_Input);
    if (d_Filter) hipFree(d_Filter);
    
    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    printf("Reset Device: ");

    hipDeviceSynchronize();
    e = hipGetLastError();
    if(e!=hipSuccess){
      printf("ERROR: %s, FILE: %s, LINE: %d\n", hipGetErrorString(e), __FILE__, __LINE__);
    }
    else{
      printf("cudaGetLastError() == cudaSuccess!\n");
    }

    hipDeviceReset();


    return 0;
}
