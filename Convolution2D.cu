/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	1000000000000
#define CHECK_CUDA_ERROR(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            goto cleanup; \
        } \
    } while (0)
 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = filterR; y < imageH+filterR; y++) {
    for (x = filterR; x < imageW+filterR; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;
        sum += h_Src[y * (imageW+2*filterR) + d] * h_Filter[filterR - k];
        h_Dst[y * (imageW+2*filterR) + x] = sum;
      }
    }
  }    
}

__global__ void convolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, 
  int imageW, int imageH, int filterR) {

  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int y = index / imageW; // row
  int x = index % imageW; // col
  y+= filterR;
  x+= filterR;
  float sum = 0;
    for(int k = -filterR; k <= filterR; k++){
      int d = x + k;
        sum += d_Src[y * (imageW+2*filterR) + d] * d_Filter[filterR - k];
    }
    d_Dst[y * (imageW + 2 * filterR) + x] = sum; 
}

__global__ void convolutionColGPU(float *d_Dst, float *d_Src, float *d_Filter, 
  int imageW, int imageH, int filterR) {
  
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int y = index / imageW; // row
  int x = index % imageW; // col
  y += filterR;
  x += filterR;
  float sum = 0;
    for(int k = -filterR; k <= filterR; k++){
      int d = y + k;
        sum += d_Src[d * (imageW+2*filterR) + x] * d_Filter[filterR - k];
    }
      d_Dst[(y-filterR) * (imageW) + (x-filterR)] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = filterR; y < imageH+filterR; y++) {
    for (x = filterR; x < imageW+filterR; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;
          sum += h_Src[d * (imageW+2*filterR) + x] * h_Filter[filterR - k];
        h_Dst[(y-filterR) * imageW + (x-filterR)] = sum;
      }
    }
  }
    
}




////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU;
    float *d_Filter = NULL;
    float*d_Input = NULL;
    float *d_Buffer = NULL;
    float *d_Output = NULL;

    int imageW;
    int imageH;
    int extra_block;  // used to indicate if an extra block is needed (if image size is not divisible by 1024)
    unsigned int i;

    // CUDA Error Checking
    hipError_t e;

    // CPU time measuring variables
    clock_t start, end;

    // CUDA measuring events
    hipEvent_t startCuda, stopCuda;
    float millisecondsTransfers = 0, millisecondsKernelsandTransferBack = 0;

	  printf("Enter filter radius : ");
    if(!scanf("%d", &filter_radius)){
      printf("ERROR: scanf: FILE: %s, LINE: %d\n", __FILE__, __LINE__);
    }

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    if(!scanf("%d", &imageW)){
      printf("ERROR: scanf: FILE: %s, LINE: %d\n", __FILE__, __LINE__);
    }
    imageH = imageW;
    extra_block = ((imageH*imageW)%1024 != 0);

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)calloc((imageW+2*filter_radius) * (imageH+2*filter_radius), sizeof(float)); // use calloc to initialize padded elements to 0
    h_Buffer    = (float *)calloc((imageW+2*filter_radius) * (imageH+2*filter_radius), sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));

    if(!(h_Filter && h_Input && h_Buffer && h_OutputCPU && h_OutputGPU)){
      printf("ERROR on Host Allocation!\n");
      goto cleanup;
    }
    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

 for (i = filter_radius; i < filter_radius + imageH; i++) {
        int row = i*(2*filter_radius + imageW);
        for(int j = filter_radius; j < imageW + filter_radius; j++){
          h_Input[row+j] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
        }
    }

    printf("Allocating GPU memory...\n");

    CHECK_CUDA_ERROR(hipMalloc((void **)&d_Filter, FILTER_LENGTH * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_Input, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_Buffer, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_Output, imageW * imageH * sizeof(float)));

    hipMemset(d_Input, 0, (imageW+2*filter_radius) * (imageH+2*filter_radius));
    hipMemset(d_Buffer, 0, (imageW+2*filter_radius) * (imageH+2*filter_radius));
    hipMemset(d_Output, 0, imageW * imageH);

    // Start Measuring memory transfer times
    hipEventCreate(&startCuda);
    hipEventCreate(&stopCuda);

    hipEventRecord(startCuda, 0);

    CHECK_CUDA_ERROR(hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_Input, h_Input, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(float), hipMemcpyHostToDevice));

    hipEventRecord(stopCuda, 0);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsTransfers, startCuda, stopCuda);

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    start = clock();
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    end = clock();
    printf("CPU time: %f msec\n", (double)((end - start)/(CLOCKS_PER_SEC/1000000))/1000);
    printf("GPU computation...\n");

    // 1st Kernel launch
    // No need for sync barrier because cudaMemCpy(..., hostToDevice) works as barrier
    //printf("1st Kernel Launch: Row Conolution: ");

    hipEventRecord(startCuda, 0 );
    convolutionRowGPU<<<(imageH*imageW)/1024+extra_block, 1024>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    e = hipGetLastError();
    if(e!=hipSuccess){
      printf("ERROR: %s, FILE: %s, LINE: %d\n", hipGetErrorString(e), __FILE__, __LINE__);
      goto cleanup;
    }

    // 2nd Kernel launch
    // Synchronize between 2 kernels launch because Column kernel needs the d_Buffer as input.
    // d_Buffer works as intermediate result so we ensure it is completely written.

    //printf("2nd Kernel Launch: Col Conolution: ");
    convolutionColGPU<<<(imageH*imageW)/1024+extra_block, 1024>>>(d_Output, d_Buffer, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
    e = hipGetLastError();
    if(e!=hipSuccess){
      printf("ERROR: %s, FILE: %s, LINE: %d\n", hipGetErrorString(e), __FILE__, __LINE__);
      goto cleanup;
    }

    CHECK_CUDA_ERROR(hipMemcpy(h_OutputGPU, d_Output, imageW * imageH * sizeof(float), hipMemcpyDeviceToHost));
    hipEventRecord(stopCuda, 0);
    hipEventSynchronize(stopCuda);
    hipEventElapsedTime(&millisecondsKernelsandTransferBack, startCuda, stopCuda);
    printf("GPU Time: %f msec\n", millisecondsTransfers+millisecondsKernelsandTransferBack);

    printf("CPU OUTPUT: \n");
      for (i = 0; i < imageW * imageH; i++) {
        printf("%f  ", h_OutputCPU[i]);
      }
    printf("\nDONE\n");
    
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  
    for (i = 0; i < imageW * imageH; i++) {
        if(ABS(h_OutputGPU[i] - h_OutputCPU[i]) > accuracy){
          printf("Difference bigger than accuray in index i = %d, h_OutputGPU[%d] = %f, h_OutputCPU[%d] = %f\n", i, i, h_OutputGPU[i], i, h_OutputCPU[i]);
          goto cleanup;
        }
    }

cleanup:
    // free all the allocated memory
    printf("Deallocating host memory\n");
    if (h_OutputCPU) free(h_OutputCPU);
    if (h_Buffer) free(h_Buffer);
    if (h_Input) free(h_Input);
    if (h_Filter) free(h_Filter);
    if (h_OutputGPU) free(h_OutputGPU);

    printf("Deallocating device memory\n");
    if (d_Output) hipFree(d_Output);
    if (d_Buffer) hipFree(d_Buffer);
    if (d_Input) hipFree(d_Input);
    if (d_Filter) hipFree(d_Filter);
    
    printf("Destroying CUDA events\n");
    CHECK_CUDA_ERROR(hipEventDestroy(startCuda));
    CHECK_CUDA_ERROR(hipEventDestroy(stopCuda));

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    printf("Reset Device: ");

    hipDeviceSynchronize();
    e = hipGetLastError();
    if(e!=hipSuccess){
      printf("ERROR: %s, FILE: %s, LINE: %d\n", hipGetErrorString(e), __FILE__, __LINE__);
    }
    else{
      printf("cudaGetLastError() == cudaSuccess!\n");
    }

    hipDeviceReset();


    return 0;
}
